#include "hip/hip_runtime.h"
// C++ Standard Libraries
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>

// C Libraries
#include <stdio.h>

// System Libraries
#include <png++/png.hpp>
#include <omp.h>
#include <hip/hip_runtime.h>

// Project Libraries
#include "ising.hpp"
#include "data.hpp"

ising::ising(std::size_t sweeps, std::size_t width, unsigned num_neighbors,
             std::size_t nimg, unsigned scale, double temp, double fstr,
             std::string backend)
{
    this->sweeps = sweeps;
    data.E.reserve(sweeps);
    data.E_fluc.reserve(sweeps);
    data.M.reserve(sweeps);
    data.M_fluc.reserve(sweeps);

    this->width = width;
    this->num_neighbors = num_neighbors;
    this->scale = scale;

    switch(num_neighbors)
    {
        case 2:
            dim = 1;
            spins = std::make_unique<char[]>(width);
            break;
        case 4:
            dim = 2;
            spins = std::make_unique<char[]>(width * width);
            break;
        case 6:
        case 8:
        case 12:
            throw std::runtime_error("Not implemented!");
            break;
        default:
            throw std::runtime_error("Option parsing missed invalid neighbor quantity!");
    }

    initialize_spins();

    this->nimg = nimg;
    this->temp = temp;
    this->field_strength = fstr;
    if (backend == "cpu-serial")
    {
        this->backend = 0;
    }
    else if (backend == "cpu-parallel")
    {
        this->backend = 1;
    }
    else if (backend == "gpu")
    {
        this->backend = 2;
    }
}

double ising::calc_deltaU(unsigned i, unsigned j)
{
    neighbors nb = get_neighbors(i, j);

    return 2 * spins[i * width + j] * (nb.top + nb.bottom + nb.left + nb.right + field_strength);
}

neighbors ising::get_neighbors(unsigned i, unsigned j)
{
    neighbors nb;

    if (i == 0)
    {
        nb.top = get_spin(width - 1, j);
        nb.bottom = get_spin(i + 1, j);

    }
    else if (i == width - 1)
    {
        nb.top = get_spin(i - 1, j);
        nb.bottom = get_spin(0, j);
    }
    else
    {
        nb.top = get_spin(i-1, j);
        nb.bottom = get_spin(i+1, j);
    }

    if (j == 0)
    {
        nb.left = get_spin(i, width - 1);
        nb.right = get_spin(i, j + 1);
    }
    else if (j == width - 1)
    {
        nb.left = get_spin(i, j - 1);
        nb.right = get_spin(i, 0);
    }
    else
    {
        nb.left = get_spin(i, j - 1);
        nb.right = get_spin(i, j + 1);
    }

    int check = nb.left * nb.right * nb.top * nb.bottom;
    check = check * check;
    if (check != 1)
    {
        std::cerr << "Error: some neighbors have garbage values! " + std::to_string(check) << std::endl;
    }
    return nb;
}

__global__ void cudaCalcTotalU(int width, int *grid, double *interactions, double h)
{
    int id = threadIdx.x;
    if (id < width * width)
    {
        int y = id / width;
        int x = id % width;

        int top, bottom, left, right;

        int spin = grid[id];

        if (y == 0)
        {
            top = grid[(width-1)*width + x];
            bottom = grid[width + x];
        }
        else if (y == width - 1)
        {
            top = grid[(y-1) * width + x];
            bottom = grid[x];
        }
        else
        {
            top = grid[(y-1)*width + x];
            bottom = grid[(y+1)*width + x];
        }

        if (x == 0)
        {
            left = grid[y*width + width-1];
            right = grid[y*width + 1];
        }
        else if (x == width - 1)
        {
            left = grid[y*width+width-2];
            right = grid[y*width];
        }
        else
        {
            left = grid[y*width + x - 1];
            right = grid[y*width + x + 1];
        }
        printf("%d %d %d %d\n", top, bottom, left, right);
        assert(top * top + bottom * bottom + left * left * right * right == 4);
        printf("%d %d %d %d\n", top, bottom, left, right);

        interactions[id] = grid[id] * (h + top + left + (y == width-1) * bottom + (x == width-1) * right);
    }

}

__global__ void reduce(double *in)
{
    int id = threadIdx.x;

    auto stride = 1;
    int threads = blockDim.x;

    while (threads > 0)
    {
        if (id < threads)
        {
            auto idx1 = id * stride * 2;
            auto idx2 = idx1 + stride;
            in[idx1] += in[idx2];
        }

        stride <<= 1;
        threads >>= 1;
    }
}

double ising::calc_totalU()
{
    auto total = 0.;

    // Our Hamiltonian is H = -epsilon*Sum[(s_i)(s_j)] - h sum[s_i]
    // So at every cell site, you sum neighbor interactions

    // Normal Stencil:
    //    |
    //  --+
    //
    //  Right Edge Stencil:
    //    |
    //  --+--
    //
    //  Bottom Edge Stencil:
    //    |
    //  --+
    //    |
    //
//    #pragma omp parallel for
    for (auto i = 0; i < width; i++)
    {
        for (auto j = 0; j < width; j++)
        {
            double interactions = 0.;
            neighbors nb = get_neighbors(i,j);

            interactions += nb.top + nb.left + field_strength;

            if (i == width-1)
            {
                interactions += nb.bottom;
            }
            if (j == width-1)
            {
                interactions += nb.right;
            }

            total -= get_spin(i,j) * interactions;
        }
    }
    return total;
}

double ising::calc_totalM()
{
    auto total = 0.;

    for (auto i = 0; i < width; i++)
    {
        for (auto j = 0; j < width; j++)
        {
            total += get_spin(i,j);
        }
    }
    return total;
}

void ising::set_display_mode(unsigned char mode)
{
    if (mode < 2)
    {
        display_mode = mode;
    }
    else
    {
        throw std::runtime_error("Invalid display mode supplied!");
    }
}

void ising::set_benchmark_mode(unsigned char mode)
{
    if (mode < 2)
    {
        benchmark_mode = mode;
    }
    else
    {
        throw std::runtime_error("Invalid benchmark mode supplied!");
    }
}

void ising::initialize_spins()
{
    std::random_device rd;
    std::mt19937 engine(rd());

    std::size_t size;

    if (dim == 1)
    {
        size = width;
    }
    else if (dim == 2)
    {
        size = width * width;
    }
    else if (dim == 3)
    {
        size = width * width * width;
        throw std::runtime_error("Dim-3 not implemented!");
    }
    else
    {
        throw std::runtime_error("Dimensionality constraint violated!");
    }

//    #pragma omp parallel for
    for (auto i = 0; i < size; i++)
    {
        spins[i] = 1 - 2 * (engine() % 2);
    }
}

void ising::run()
{
    std::random_device rd;
    std::mt19937 engine(rd());
    std::uniform_int_distribution<int> dist(0, width - 1);
    std::uniform_real_distribution<double> floatdist;
    // We want to print `nimg` images, so we print every `iter/nimg` step. But
    // this isn't always an integer, so let's increase iter until it is.
    for (auto T = 0.; T < 6; T += 0.01)
    {
        initialize_spins();
        auto iters = sweeps * width;

        double avg_E = 0.;
        double avg_E_square = 0.;
        double avg_M = 0.;
        double avg_M_square = 0.;

        data.T.push_back(T);
        std::cerr << "Performing " << iters << " iterations...\n";
        for (auto t = 0; t < iters; t++)
        {
            auto i = dist(engine);
            auto j = dist(engine);
            auto dU = calc_deltaU(i, j);

            if (dU <= 0.)
            {
                //auto u1 = calc_totalU();
                flip_spin(i,j);
                //auto u2 = calc_totalU();

                //auto totald = (u2-u1) - dU;
                // TODO: diagnose why we are getting errors as bad as 2...
                //if (totald > 4)
                {
                //    throw std::runtime_error("Energy " + std::to_string(totald) + " doesn't match!");
                }
            }
            else
            {
                // If T approaches zero, then the boltzmann factor becomes infinitely small
                if (T != 0.)
                {
                    if (floatdist(engine) < std::exp(-dU / T))
                    {
                        flip_spin(i,j);
                    }
                }
            }

            // Equilibrium sweeps contribute to equilibirum averages
            if (t > 10*width*width)
            {
                hipError_t err;
                int *d_spins;
                double *d_interactions;
                err = hipMalloc(&d_spins, width*width*sizeof(int));
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                err =hipMalloc(&d_interactions, width*width*sizeof(double));
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                err =hipMemcpy(d_spins, spins.get(), width*sizeof(int), hipMemcpyHostToDevice);
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                err =hipDeviceSynchronize();
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                cudaCalcTotalU<<<1, width*width>>>(width, d_spins, d_interactions, field_strength);
                err =hipDeviceSynchronize();
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                // reduce<<<1, width*width>>>(d_interactions);
                err = hipDeviceSynchronize();
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                double E;
                err = hipMemcpy(&E, d_interactions, sizeof(double), hipMemcpyDeviceToHost);
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                err =hipFree(d_spins);
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                err =hipFree(d_interactions);
                if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
                // std::cerr << "E has value " << E << std::endl;
                avg_E += E;
                avg_E_square += E * E;
                auto M = calc_totalM();
                avg_M += M;
                avg_M_square += M * M;
            }

            /* Now, handle visualization
            if ((t+1) % width == 0)
            {
                data.E.push_back(calc_totalU());
                if (display_mode == 0)
                {
                    print_snapshot();
                }
                else
                {
                    char filename[256];
                    snprintf(filename, 256, "snapshot_%lux%lu_%.8lu.png", width, width, t / width);
                    save_png_snapshot(filename);
                }
                std::cerr << "Total energy at sweep " << t / width << " is " << data.E.at(t / width) << std::endl;
            }
            */
        }

        auto count = iters - 10 * width * width;
        avg_E /= count;
        avg_E_square /= count;
        avg_M /= count;
        avg_M_square /= count;

        data.E.push_back(avg_E);
        data.E_fluc.push_back(avg_E_square - avg_E * avg_E);
        data.M.push_back(avg_M);
        data.M_fluc.push_back(avg_M_square - avg_M * avg_M);
        std::cerr << "Equilibrium E: " << data.E.back() << std::endl;
    }

    std::ofstream f1("output_" + std::to_string(width) + ".dat");

    for (auto i = 0; i < data.T.size(); i++)
    {
        f1 << static_cast<double>(i)*0.05 << " " << data.E[i] << " " << data.E_fluc[i] << " " << data.M[i] << " " << data.M_fluc[i] << "\n";
    }
}

inline void ising::flip_spin(std::size_t i, std::size_t j)
{
    auto spin = get_spin(i,j);

    if (spin * spin != 1)
    {
        throw std::runtime_error("Invalid spin detected");
    }
    spins[i * width + j] = -spin;
}

int ising::get_spin(std::size_t i, std::size_t j) const
{
    return spins[i * width + j];
}

void ising::save_png_snapshot(const char* fname)
{
    // TODO: generalize to more dims
    png::image<png::gray_pixel_1> img(scale * width, scale * width);
    for (auto y = 0; y < width; ++y)
    {
        for (auto i = 0; i < scale; i++)
        {
            for (auto x = 0; x < width; x++)
            {
                for (auto j = 0; j < scale; j++)
                {
                    img[scale*y + i][scale*x + j] = png::gray_pixel_1(get_spin(y,x) > 0);
                }
            }
        }
    }
    img.write(fname);
}

void ising::print_snapshot()
{
    for (auto y = 0; y < width; ++y)
    {
        for (auto i = 0; i < scale; i++)
        {
            for (auto x = 0; x < width; x++)
            {
                for (auto j = 0; j < scale; j++)
                {
                    std::cout << (get_spin(y,x) > 0 ? '.' : '+');
                }
            }
            std::cout << '\n';
        }
    }
    std::cout << "\n\n";
}
